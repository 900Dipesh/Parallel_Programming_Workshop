
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#include<sys/time.h>

#define N 999999999
#define THDS_PER_BLK	256
/*
	for(i=0;i<N;i++)
	{
		y = sqrt(1-x*x);
		area += y*dx;  
		x += dx;
	}
*/

void my_cudasafe( hipError_t error, char* message)
{
	if(error!=hipSuccess) 
	{ 
		fprintf(stderr,"ERROR: %s : %s\n",message,hipGetErrorString(error)); 
		exit(-1); 
	}
}

__global__ void arradd(double* global_area)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;
	int i;
	double x, y, dx, area, tmp;
	dx = 1.0/N;
	area = 0.0;
	__shared__ double tmp_area[THDS_PER_BLK];
	if(myid < N)
	{
		x = myid*dx;
		y = sqrt(1-x*x);
		tmp_area[threadIdx.x] = y*dx;
		
		__syncthreads();
		
		if(threadIdx.x == 0)
		{
			for(i=0;i<THDS_PER_BLK;i++)
			{
				area += tmp_area[i]; 
			}
			tmp = atomicAdd(&global_area[0],area);
		}
	}
}


int main()
{
	int i=0;
		
	double *area, pi, *area_d;
	double exe_time;
	struct timeval stop_time, start_time;

	gettimeofday(&start_time, NULL);
	
	area = (double *)malloc(sizeof(double));
 	my_cudasafe(hipMalloc((void**)&area_d, sizeof(double)), "cuda Malloc : area_d");

	int num_blocks  = (N / THDS_PER_BLK)+1;

	arradd<<< num_blocks,THDS_PER_BLK >>>(area_d);
	my_cudasafe(hipGetLastError(), "arradd kernel");

	my_cudasafe(hipMemcpy(area, area_d, 1*sizeof(double), hipMemcpyDeviceToHost),"cuda Memcpy : area_d DTH");
	

	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	pi = 4.0*area[0];
	printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);

	free(area);
	hipFree(area_d);
}
